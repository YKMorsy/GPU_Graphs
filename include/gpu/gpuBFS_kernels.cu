#include "hip/hip_runtime.h"
#include "gpuBFS_kernels.cuh"
#include <stdio.h>



__global__
void init_distance_kernel(int size, int *device_distance, int source)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < size)
    {
        device_distance[i] = INF;
    }

    if (i == source)
    {
        device_distance[i] = 0;
    }
}

__global__ 
void linear_bfs(const int num_nodes, const int* row_offset, const int* column_index, int* distance, const int iteration, const int* in_queue, const int in_queue_count, int* out_queue, int* out_queue_count, int* edges_traversed)
{
	// Compute index of corresponding vertex in the queue.
	int global_tid = blockIdx.x * blockDim.x + threadIdx.x;

	do
	{
		// skip thread if nothing for it to process
		if(global_tid >= in_queue_count) continue;

		int local_edges_traversed = 0;

		// Get node from the queue
		int v = in_queue[global_tid];

		// get neighbors range from offset
		int r = row_offset[v];
		int r_end = row_offset[v+1];

		for(int offset = r; offset < r_end; offset++)
		{
			// get neighbor
			int j = column_index[offset];

			// add neighbor if not traversed
			if(distance[j] == INF)
			{
				local_edges_traversed+=1;
				distance[j]=iteration+1;
				// Enqueue vertex.
				int ind = atomicAdd(out_queue_count,1);
				out_queue[ind]=j;
			}
		}
		global_tid += gridDim.x*blockDim.x;
		atomicAdd(edges_traversed, local_edges_traversed);
	} 
	// ensure atleast one thread has something to process
	while(__syncthreads_or(global_tid < in_queue_count)); 
}

 __device__ int warp_cull(volatile int scratch[WARPS][HASH_RANGE], const int v)
{
	//unsigned int active = __ballot_sync(FULL_MASK, v >= 0);
	//if( v == -1) return v;
	const int hash = v & (HASH_RANGE-1);
	const int warp_id = threadIdx.x / WARP_SIZE;
	if(v >= 0)
		scratch[warp_id][hash]= v;
	__syncwarp();
	const int retrieved = v >= 0 ? scratch[warp_id][hash] : v;
	__syncwarp();
	unsigned int active = __ballot_sync(FULL_MASK, retrieved == v);
	if (retrieved == v)
	{
		// Vie to be the only thread in warp inspecting vertex v.
		scratch[warp_id][hash] = threadIdx.x;
		__syncwarp(active);
		// Some other thread has this vertex
		if(scratch[warp_id][hash] != threadIdx.x)
			return -1;
	}
	return v;
}

__device__ 
prescan_result block_prefix_sum(const int val)
{
	// Heavily inspired/copied from sample "shfl_scan" provided by NVIDIA.
	// Block-wide prefix sum using shfl intrinsic.
	volatile __shared__ int sums[WARPS];
	int value = val;

	const int lane_id = threadIdx.x % WARP_SIZE;
	const int warp_id = threadIdx.x / WARP_SIZE;

	// Warp-wide prefix sums.
#pragma unroll
	for(int i = 1; i <= WARP_SIZE; i <<= 1)
	{
		const int n = __shfl_up_sync(FULL_MASK, value, i, WARP_SIZE);
		if (lane_id >= i)
			value += n;
	}

	// Write warp total to shared array.
	if (lane_id == WARP_SIZE- 1)
	{
		sums[warp_id] = value;
	}

	__syncthreads();

	// Prefix sum of warp sums.
	if (warp_id == 0 && lane_id < WARPS)
	{
		int warp_sum = sums[lane_id];
		const unsigned int mask = (1 << (WARPS)) - 1;
#pragma unroll
		for (int i = 1; i <= WARPS; i <<= 1)
		{
			const int n = __shfl_up_sync(mask, warp_sum, i, WARPS);
			if (lane_id >= i)
				warp_sum += n;
		}

		sums[lane_id] = warp_sum;
	}

	__syncthreads();

	// Add total sum of previous warps to current element.
	if (warp_id > 0)
	{
		const int block_sum = sums[warp_id-1];
		value += block_sum;
	}

	// printf("sum %d\n", sums[warp_id-1]);

	prescan_result result;
	// Subtract value given by thread to get exclusive prefix sum.
	result.offset = value - val;
	// Get total sum.
	result.total = sums[WARPS-1];
	return result; 
}

__device__ 
void block_gather(const int* column_index, int* distance, const int iteration, int* out_queue, int* out_queue_count, int r, int r_end)
{
	volatile __shared__ int comm[3];
	while(__syncthreads_or(r < r_end))
	{
		// Vie for control of block.
		if(r < r_end)
			comm[0] = threadIdx.x;
		__syncthreads();
		if(comm[0] == threadIdx.x)
		{
			// If won, share your range to the entire block.
			comm[1] = r;
			comm[2] = r_end;
			r = r_end;
		}
		__syncthreads();
		int r_gather = comm[1] + threadIdx.x;
		const int r_gather_end = comm[2];
		const int total = comm[2] - comm[1];
		int block_progress = 0;

		// entire block gets the neighbors of one thread's nodes
		while((total - block_progress) > 0)
		{
			int neighbor = -1;
			bool is_valid = false;
			if (r_gather < r_gather_end)
			{
				neighbor = column_index[r_gather];
				// Look up status of current neighbor.
				if(distance[neighbor] == INF)
				{
					is_valid = true;
					// Update label.
					distance[neighbor] = iteration + 1;
				}
			}
			// Obtain offset in queue by computing prefix sum
			const prescan_result prescan = block_prefix_sum(is_valid?1:0);
			volatile __shared__ int base_offset[1];

			// Obtain base enqueue offset and share it to whole block.
			if(threadIdx.x == 0)
				base_offset[0] = atomicAdd(out_queue_count,prescan.total);
			__syncthreads();
			// Write vertex to the out queue.
			if (is_valid)
				out_queue[base_offset[0]+prescan.offset] = neighbor;

			r_gather += 1024;
			block_progress+= 1024;
			__syncthreads();
		}
	}
}


__device__ 
void fine_gather(const int* column_index, int* distance, const int iteration, int* out_queue, int* out_queue_count, int r, int r_end)
{
	prescan_result rank = block_prefix_sum(r_end-r);

	// printf("%d\n", rank.total);
	// printf("%d\n", r);
	// printf("%d\n\n", r_end);

	__shared__ int comm[1024];
	int cta_progress = 0;

	while ((rank.total - cta_progress) > 0)
	{
		// Pack shared array with neighbors from adjacency lists.
		while((rank.offset < cta_progress + 1024) && (r < r_end))
		{
			comm[rank.offset - cta_progress] = r;
			rank.offset++;
			r++;
		}
		__syncthreads();
		// label neighbor distance
		int neighbor;
		bool is_valid = false;
		if (threadIdx.x < (rank.total - cta_progress))
		{
			neighbor = column_index[comm[threadIdx.x]];
			if(distance[neighbor] == INF)
			{
				is_valid = true;
				// Update label
				distance[neighbor] = iteration + 1;
			}
		}
		__syncthreads();

		// add neighbor to enqueue

		// Obtain offset in queue by computing prefix sum.
		const prescan_result prescan = block_prefix_sum(is_valid?1:0);
		volatile __shared__ int base_offset[1];
		// Obtain base enqueue offset
		if(threadIdx.x == 0)
		{
			base_offset[0] = atomicAdd(out_queue_count,prescan.total);
		}
		__syncthreads();
		const int queue_index = base_offset[0] + prescan.offset;
		// Write to queue
		if (is_valid)
		{
			out_queue[queue_index] = neighbor;
		}

		cta_progress += 1024;
		__syncthreads();
	}
}



__global__ 
void expand_contract_bfs(const int num_nodes, const int* row_offset, const int* column_index, int* distance, const int iteration, const int* in_queue, const int in_queue_count, int* out_queue, int* out_queue_count)
{
	// Compute index of corresponding vertex in the queue.
	int global_tid = blockIdx.x * blockDim.x + threadIdx.x;

	do
	{
		// Get node from the queue
		int v = global_tid < in_queue_count? in_queue[global_tid]:-1;

		// Do local warp-culling.
		volatile __shared__ int scratch[WARPS][HASH_RANGE];
		v = warp_cull(scratch, v);

		// get neighbors range from offset
		int r = row_offset[v];
		int r_end = row_offset[v+1];
		bool big_list = (r_end - r) >= 1024;

		block_gather(column_index, distance, iteration, out_queue, out_queue_count, r, big_list ? r_end : r);
		fine_gather(column_index, distance, iteration, out_queue, out_queue_count, r, big_list ? r : r_end);

		global_tid += gridDim.x*blockDim.x;
	} 
	// ensure atleast one thread has something to process
	while(__syncthreads_or(global_tid < in_queue_count)); 
}
