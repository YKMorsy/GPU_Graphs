#include "hip/hip_runtime.h"
#include "gpuBFS.cuh"

__global__
void nextLayer(int level, int *d_adjacencyList, int *d_edgesOffset, int *d_distance, int *d_parent,
               int queueSize, int *d_currentQueue) 
{
    int thid = blockIdx.x * blockDim.x + threadIdx.x;

    if (thid < queueSize) 
    {
        int cur_node = d_currentQueue[thid];
        int row_offset_start = cur_node < 0 ? 0 : d_edgesOffset[cur_node];
        int row_offset_end = cur_node < 0 ? 0 : d_edgesOffset[cur_node+1];
        for (int i = row_offset_start; i < row_offset_end; i++) {
            int v = d_adjacencyList[i];
            if (d_distance[v] == -1) 
            {
                // printf("neighbor %d\n", v);
                d_distance[v] = level + 1;
                d_parent[v] = i;
            }
        }
    }
}

__global__
void countDegrees(int *d_adjacencyList, int *d_edgesOffset, int *d_parent,
                  int queueSize, int *d_currentQueue, int *d_degrees)
{
    int thid = blockIdx.x * blockDim.x + threadIdx.x;

    if (thid < queueSize) {
        int cur_node = d_currentQueue[thid];
        int row_offset_start = cur_node < 0 ? 0 : d_edgesOffset[cur_node];
        int row_offset_end = cur_node < 0 ? 0 : d_edgesOffset[cur_node+1];

        int degree = 0;

        for (int i = row_offset_start; i < row_offset_end; i++) {
            int v = d_adjacencyList[i];
            if (d_parent[v] == i && v != cur_node) 
            {
                degree++;
            }
        }
        d_degrees[thid] = degree;
    }
}

__global__
void block_prefix_sum(int size, int *d_degrees, int *incrDegrees) 
{
    int thid = blockIdx.x * blockDim.x + threadIdx.x;

    if (thid < size) {
        //write initial values to shared memory
        __shared__ int prefixSum[1024];
        int modulo = threadIdx.x;
        prefixSum[modulo] = d_degrees[thid];
        __syncthreads();

        //calculate scan on this block
        //go up
        for (int nodeSize = 2; nodeSize <= 1024; nodeSize <<= 1) {
            if ((modulo & (nodeSize - 1)) == 0) {
                if (thid + (nodeSize >> 1) < size) {
                    int nextPosition = modulo + (nodeSize >> 1);
                    prefixSum[modulo] += prefixSum[nextPosition];
                }
            }
            __syncthreads();
        }

        //write information for increment prefix sums
        if (modulo == 0) {
            int block = thid >> 10;
            incrDegrees[block + 1] = prefixSum[modulo];
            // printf("total %d %d\n", block + 1, incrDegrees[block + 1]);
        }

        //go down
        for (int nodeSize = 1024; nodeSize > 1; nodeSize >>= 1) {
            if ((modulo & (nodeSize - 1)) == 0) {
                if (thid + (nodeSize >> 1) < size) {
                    int next_position = modulo + (nodeSize >> 1);
                    int tmp = prefixSum[modulo];
                    prefixSum[modulo] -= prefixSum[next_position];
                    prefixSum[next_position] = tmp;

                }
            }
            __syncthreads();
        }
        d_degrees[thid] = prefixSum[modulo];

        // printf("individiual %d %d\n", thid, d_degrees[thid]);
    }

}
__global__
void gather(int *d_adjacencyList, int *d_edgesOffset, int *d_parent, int queueSize,
                             int *d_currentQueue, int *d_nextQueue, int *d_degrees, int *incrDegrees)
{
    int thid = blockIdx.x * blockDim.x + threadIdx.x;

    if (thid < queueSize) {
        __shared__ int sharedIncrement;
        if (!threadIdx.x) {
            sharedIncrement = incrDegrees[thid >> 10];
        }
        __syncthreads();

        int sum = 0;
        if (threadIdx.x) {
            sum = d_degrees[thid - 1];
        }

        int cur_node = d_currentQueue[thid];
        int row_offset_start = cur_node < 0 ? 0 : d_edgesOffset[cur_node];
        int row_offset_end = cur_node < 0 ? 0 : d_edgesOffset[cur_node+1];
        int counter = 0;
        for (int i = row_offset_start; i < row_offset_end; i++)
        {
            int v = d_adjacencyList[i];
            if (d_parent[v] == i && v != cur_node) {
                int nextQueuePlace = sharedIncrement + sum + counter;
                // printf("individiual %d %d\n", thid, nextQueuePlace);
                d_nextQueue[nextQueuePlace] = v;
                counter++;
            }
        }
    }
}

__global__
void init_distance_kernel(int *device_distance, int *device_parent, int size, int source)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size)
    {
        if (i == source)
        {
            device_distance[i] = 0;
            device_parent[i] = 0;
        }
        else
        {
            device_distance[i] = -1;
            device_parent[i] = -1;
        }

    }
}

__host__
gpuBFS::gpuBFS(csr &graph, int source)
{
    hipMalloc(&d_col_idx, graph.num_edges * sizeof(int));
    hipMalloc(&d_row_offset, (graph.num_nodes+1) * sizeof(int));
    // hipMalloc(&d_edges_size, graph.num_nodes * sizeof(int));
    hipMalloc(&d_distance, graph.num_nodes * sizeof(int));
    hipMalloc(&d_parent, graph.num_nodes * sizeof(int));
    hipMalloc(&d_in_q, graph.num_nodes * sizeof(int));
    hipMalloc(&d_out_q, graph.num_nodes * sizeof(int));
    hipMalloc(&d_degrees, graph.num_nodes * sizeof(int));

    hipHostMalloc((void **) &d_degrees_total, graph.num_nodes * sizeof(int));

    init_distance(graph, source);

    int firstElementQueue = source;
    hipMemcpy(d_in_q, &firstElementQueue, sizeof(int), hipMemcpyHostToDevice);
    
    // d_degrees_total = (int *)malloc(graph.num_nodes * sizeof(int));

    hipMemcpy(d_col_idx, graph.col_idx, graph.num_edges * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_row_offset, graph.row_offset, (graph.num_nodes+1) * sizeof(int), hipMemcpyHostToDevice);

    hipEvent_t gpu_start, gpu_end;
    hipEventCreate(&gpu_start);
    hipEventCreate(&gpu_end);
    hipEventRecord(gpu_start);

    int *queueSize;
    queueSize = (int *)malloc(sizeof(int));
    *queueSize = 1;
    int *nextQueueSize;
    nextQueueSize = (int *)malloc(sizeof(int));
    *nextQueueSize = 0;
    iteration = 0;
    while (*queueSize)
    {

        // std::cout << "iter and size: " << iteration << " " << *queueSize << std::endl;
        
        // next layer phase
        int block_size = 1024;
        int num_blocks = *queueSize / block_size + 1;

        // std::cout << num_blocks << std::endl;
        
        nextLayer<<<num_blocks, block_size>>>(iteration, d_col_idx, d_row_offset, d_distance, d_parent, *queueSize, d_in_q);
        hipDeviceSynchronize();

        // hipMemcpy(host_distance, d_distance, graph.num_nodes * sizeof(int), hipMemcpyDeviceToHost);

        // std::cout << host_distance[1] << " " << host_distance[2] << std::endl;

        countDegrees<<<num_blocks, block_size>>>(d_col_idx, d_row_offset, d_parent, *queueSize, d_in_q, d_degrees);
        hipDeviceSynchronize();
        
        block_prefix_sum<<<num_blocks, block_size>>>(*queueSize, d_degrees, d_degrees_total) ;
        hipDeviceSynchronize();

        *nextQueueSize = d_degrees_total[(*queueSize - 1) / 1024 + 1];

        // std::cout << *nextQueueSize << std::endl;

        gather<<<num_blocks, block_size>>>(d_col_idx, d_row_offset, d_parent, *queueSize, d_in_q, d_out_q, d_degrees, d_degrees_total);
        hipDeviceSynchronize();

        iteration++;
        *queueSize = *nextQueueSize;
        std::swap(d_in_q, d_out_q);

        hipDeviceSynchronize();
    }

    hipMemcpy(host_distance, d_distance, graph.num_nodes * sizeof(int), hipMemcpyDeviceToHost);
    // std::cout << host_distance[1] << " " << host_distance[2] << std::endl;
    // host_distance[source] = 0;

    hipEventRecord(gpu_end);
    hipEventSynchronize(gpu_end);
    hipEventElapsedTime(&exec_time, gpu_start, gpu_end);

    free(queueSize);
    free(nextQueueSize);

}

__host__
void gpuBFS::init_distance(csr &graph, int source)
{
    // allocate host memory
    host_distance = (int *)malloc(graph.num_nodes * sizeof(int));

    // allocate device memory
    hipMalloc(&d_distance, graph.num_nodes * sizeof(int));

    // copy memory from host to device
    hipMemcpy(d_distance, host_distance, graph.num_nodes * sizeof(int), hipMemcpyHostToDevice);

    // run kernel to inialize distance
    dim3 block(1024, 1);
    dim3 grid((graph.num_nodes+block.x-1)/block.x, 1);
    init_distance_kernel<<<grid, block>>>(d_distance, d_parent, graph.num_nodes, source);

    // copy back
    hipMemcpy(host_distance, d_distance, graph.num_nodes * sizeof(int), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
}


__host__
gpuBFS::~gpuBFS()
{
    free(host_distance);
    // free(host_queue);
    

    hipFree(d_distance);
    hipFree(d_in_q);
    hipFree(d_out_q);
    hipFree(d_parent);
    hipFree(d_degrees);
    hipFree(d_col_idx);
    hipFree(d_row_offset);
    hipFree(d_degrees_total);
}
