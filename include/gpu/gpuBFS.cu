#include "hip/hip_runtime.h"
#include "gpuBFS.cuh"

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "'" << std::endl;
        hipDeviceReset(); // Make sure we call CUDA Device Reset before exiting
        exit(99);
    }
}

__global__
void nextLayer(int *d_adjacencyList, int *d_edgesOffset, int *d_parent,
                int queueSize, int *d_currentQueue, int *d_distance, int iteration) {
    int thid = blockIdx.x * blockDim.x + threadIdx.x;

    if (thid < queueSize) {
        int cur_node = d_currentQueue[thid];
        int row_offset_start = d_edgesOffset[cur_node];
        int row_offset_end = d_edgesOffset[cur_node+1];

        for (int i = row_offset_start; i < row_offset_end; i++) {
            int v = d_adjacencyList[i];
            if (d_distance[v] == -1) 
            {
                d_parent[v] = i;
                d_distance[v] = iteration + 1;
            }
        }
    }
}

__global__
void countDegrees(int *d_adjacencyList, int *d_edgesOffset, int *d_parent,
                  int queueSize, int *d_currentQueue, int *d_degrees, int *d_distance, int iteration)
{
    int thid = blockIdx.x * blockDim.x + threadIdx.x;

    if (thid < queueSize) {
        int cur_node = d_currentQueue[thid];
        int row_offset_start = d_edgesOffset[cur_node];
        int row_offset_end = d_edgesOffset[cur_node+1];

        int degree = 0;

        for (int i = row_offset_start; i < row_offset_end; i++) {
            int v = d_adjacencyList[i];
            if (d_parent[v] == i && v != cur_node) 
            {
                degree++;
            }
        }

        d_degrees[thid] = degree;
    }
}

__global__
void gather(int *d_adjacencyList, int *d_edgesOffset, int *d_parent, int queueSize,
            int *d_currentQueue, int *d_nextQueue, int *incrDegrees, int *d_distance, int iteration)
{
    int thid = blockIdx.x * blockDim.x + threadIdx.x;

    if (thid < queueSize) {
        int nextQueuePlace = incrDegrees[thid];

        int cur_node = d_currentQueue[thid];
        int row_offset_start = d_edgesOffset[cur_node];
        int row_offset_end = d_edgesOffset[cur_node+1];
        for (int i = row_offset_start; i < row_offset_end; i++)
        {
            int v = d_adjacencyList[i];
            if (d_parent[v] == i && v != cur_node) 
            {
                d_nextQueue[nextQueuePlace] = v;
                nextQueuePlace++;
            }
        }
    }
}

__global__
void init_distance_kernel(int *device_distance, int *device_parent, int size, int source)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size)
    {
        if (i == source)
        {
            device_distance[i] = 0;
            device_parent[i] = -1;
        }
        else
        {
            device_distance[i] = -1;
            device_parent[i] = -1;
        }

    }
}

__host__
gpuBFS::gpuBFS(csr &graph, int source)
{    
    init_device(graph, source);

    PrefixSum ps;

    hipEvent_t gpu_start, gpu_end;
    hipEventCreate(&gpu_start);
    hipEventCreate(&gpu_end);
    hipEventRecord(gpu_start);

    int *queueSize;
    queueSize = (int *)malloc(sizeof(int));
    *queueSize = 1;
    int *nextQueueSize;
    nextQueueSize = (int *)malloc(sizeof(int));
    *nextQueueSize = 0;
    iteration = 0;

    int block = 1024;
    while (*queueSize)
    {
        std::cout << "iter and size " << iteration << " " << *queueSize << std::endl;

        int grid = (*queueSize+block-1)/block;

        nextLayer<<<grid, block>>>(d_col_idx, d_row_offset, d_parent, *queueSize, d_in_q, d_distance, iteration);
        // hipDeviceSynchronize();

        countDegrees<<<grid, block>>>(d_col_idx, d_row_offset, d_parent, *queueSize, d_in_q, d_degrees, d_distance, iteration);
        // hipDeviceSynchronize();

        ps.sum_scan_blelloch(d_degrees_total, d_degrees, *queueSize+1);
        // hipDeviceSynchronize();

        hipMemcpy(nextQueueSize, &d_degrees_total[*queueSize], sizeof(int), hipMemcpyDeviceToHost);

        gather<<<grid, block>>>(d_col_idx, d_row_offset, d_parent, *queueSize, d_in_q, d_out_q, d_degrees_total, d_distance, iteration);
        // hipDeviceSynchronize();

        iteration++;
        *queueSize = *nextQueueSize;
        int *temp = d_in_q;
        d_in_q = d_out_q;
        d_out_q = temp;
    }

    hipMemcpy(host_distance, d_distance, graph.num_nodes * sizeof(int), hipMemcpyDeviceToHost);

    hipEventRecord(gpu_end);
    hipEventSynchronize(gpu_end);
    hipEventElapsedTime(&exec_time, gpu_start, gpu_end);

    free(queueSize);
    free(nextQueueSize);
}

__host__
void gpuBFS::init_device(csr &graph, int source)
{
    host_distance = (int *)malloc(graph.num_nodes * sizeof(int));

    checkCudaErrors(hipMalloc(&d_distance, graph.num_nodes * sizeof(int)));
    checkCudaErrors(hipMalloc(&d_col_idx, graph.num_edges * sizeof(int)));
    checkCudaErrors(hipMalloc(&d_row_offset, (graph.num_nodes+1) * sizeof(int)));
    checkCudaErrors(hipMalloc(&d_distance, graph.num_nodes * sizeof(int)));
    checkCudaErrors(hipMalloc(&d_parent, graph.num_nodes * sizeof(int)));
    checkCudaErrors(hipMalloc(&d_in_q, graph.num_nodes * sizeof(int)));
    checkCudaErrors(hipMalloc(&d_out_q, graph.num_nodes * sizeof(int)));

    checkCudaErrors(hipMalloc(&d_degrees, graph.num_nodes * sizeof(int)));
    checkCudaErrors(hipMalloc(&d_degrees_total, graph.num_nodes * sizeof(int)));

    dim3 block(1024, 1);
    dim3 grid((graph.num_nodes+block.x-1)/block.x, 1);
    init_distance_kernel<<<grid, block>>>(d_distance, d_parent, graph.num_nodes, source);
    hipDeviceSynchronize();

    int firstElementQueue = source;
    hipMemcpy(d_in_q, &firstElementQueue, sizeof(int), hipMemcpyHostToDevice);

    checkCudaErrors(hipMemcpy(d_col_idx, graph.col_idx, graph.num_edges * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_row_offset, graph.row_offset, (graph.num_nodes+1) * sizeof(int), hipMemcpyHostToDevice));
}

__host__
gpuBFS::~gpuBFS()
{
    free(host_distance);    

    hipFree(d_distance);
    hipFree(d_in_q);
    hipFree(d_out_q);
    hipFree(d_parent);
    hipFree(d_degrees);
    hipFree(d_col_idx);
    hipFree(d_row_offset);
    hipFree(d_degrees_total);
}
