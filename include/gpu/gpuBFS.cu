#include "gpuBFS.cuh"

gpuBFS::gpuBFS(csr &graph, int source) 
{
    // init queue
    hipMalloc(&d_in_q, graph.num_nodes * sizeof(int));
    hipMalloc(&d_out_q, graph.num_nodes * sizeof(int));
    hipMalloc(&d_q_count, sizeof(int));

    h_q_count = 1;
    hipMemcpy(d_in_q, &source, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_q_count, &h_q_count, sizeof(int), hipMemcpyHostToDevice);

    // init distance
    host_distance = (int *)malloc(graph.num_nodes * sizeof(int));
    hipMalloc(&d_distance, graph.num_nodes * sizeof(int));
    init_distance_kernel<<< (graph.num_nodes+1024-1)/1024, 1024 >>>(graph.num_nodes, d_distance, source);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }

    // init graph
    hipMalloc(&d_row_offset, (graph.num_nodes+1) * sizeof(int));
    hipMalloc(&d_col_idx, graph.num_edges * sizeof(int));
    hipMemcpy(d_row_offset, graph.row_offset, (graph.num_nodes+1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col_idx, graph.col_idx, graph.num_edges * sizeof(int), hipMemcpyHostToDevice);
    
    hipEvent_t gpu_start, gpu_end;
    hipEventCreate(&gpu_start);
    hipEventCreate(&gpu_end);
    hipEventRecord(gpu_start);
    
    total_edges_traversed = 0;
    int *d_edges_traversed;
    hipMalloc(&d_edges_traversed, sizeof(int));
    hipMemset(d_edges_traversed, 0, sizeof(int));

    hipDeviceSynchronize();

    int iteration = 0;
    while(h_q_count > 0)
    {
        hipMemset(d_q_count,0,sizeof(int));
        // linear_bfs<<< (h_q_count+1024-1)/1024, 1024 >>>(graph.num_nodes, d_row_offset, d_col_idx, d_distance, iteration, d_in_q, h_q_count, d_out_q, d_q_count, d_edges_traversed);
        expand_contract_bfs<<< (h_q_count+1024-1)/1024, 1024 >>>(graph.num_nodes, d_row_offset, d_col_idx, d_distance, iteration, d_in_q, h_q_count, d_out_q, d_q_count);
        hipMemcpy(&h_q_count, d_q_count, sizeof(int), hipMemcpyDeviceToHost);
        std::swap(d_in_q,d_out_q);
        iteration++;
    }

    hipMemcpy(host_distance, d_distance, graph.num_nodes * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&total_edges_traversed, d_edges_traversed, sizeof(int), hipMemcpyDeviceToHost);

    hipEventRecord(gpu_end);
    hipEventSynchronize(gpu_end);
    hipEventElapsedTime(&exec_time, gpu_start, gpu_end);
}

gpuBFS::~gpuBFS() 
{
    free(host_distance);

    hipFree(d_row_offset);
    hipFree(d_col_idx);

    hipFree(d_distance);
    hipFree(d_in_q);
    hipFree(d_out_q);
    hipFree(d_q_count);
}

void gpuBFS::print_distance(csr &graph)
{
    std::cout << "\n------GPU DISTANCE VECTOR------" << std::endl;

    for (long long int i = 0; i < graph.num_nodes; i++) 
    {
        std::cout << host_distance[i] << " | ";
    }
    std::cout << std::endl;
}

