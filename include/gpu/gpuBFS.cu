#include "hip/hip_runtime.h"
#include "gpuBFS.cuh"

__global__
void countDegrees(int *d_adjacencyList, int *d_edgesOffset, int *d_parent,
                  int queueSize, int *d_currentQueue, long long int *d_degrees, int *d_distance)
{
    int thid = blockIdx.x * blockDim.x + threadIdx.x;

    if (thid < queueSize) {
        int cur_node = d_currentQueue[thid];
        int row_offset_start = d_edgesOffset[cur_node];
        int row_offset_end = d_edgesOffset[cur_node+1];

        int degree = 0;

        // printf("thid and start and end %lld %lld %lld\n", thid, row_offset_start, row_offset_end);

        for (int i = row_offset_start; i < row_offset_end; i++) {
            int v = d_adjacencyList[i];
            if (d_distance[v] == -1) 
            {
                degree++;
            }
        }
        d_degrees[thid] = degree;
    }
}

__global__ void prescan_large_unoptimized(int *output, int *input, int n, int *sums) {
	int blockID = blockIdx.x;
	int threadID = threadIdx.x;
	int blockOffset = blockID * n;

	extern __shared__ int temp[];
	temp[2 * threadID] = input[blockOffset + (2 * threadID)];
	temp[2 * threadID + 1] = input[blockOffset + (2 * threadID) + 1];

	int offset = 1;
	for (int d = n >> 1; d > 0; d >>= 1) // build sum in place up the tree
	{
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			temp[bi] += temp[ai];
		}
		offset *= 2;
	}
	__syncthreads();


	if (threadID == 0) {
		sums[blockID] = temp[n - 1];
		temp[n - 1] = 0;
	}

	for (int d = 1; d < n; d *= 2) // traverse down tree & build scan
	{
		offset >>= 1;
		__syncthreads();
		if (threadID < d)
		{
			int ai = offset * (2 * threadID + 1) - 1;
			int bi = offset * (2 * threadID + 2) - 1;
			int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}
	__syncthreads();

	output[blockOffset + (2 * threadID)] = temp[2 * threadID];
	output[blockOffset + (2 * threadID) + 1] = temp[2 * threadID + 1];
}


// __global__
// void block_prefix_sum(int size, long long int *d_degrees) 
// {
//     int thid = blockIdx.x * blockDim.x + threadIdx.x;
//     if (thid <= size) 
//     {
//         __shared__ long long block_data[1024]; // Assuming maximum block size of 1024 threads

//         int thid_block = threadIdx.x;
//         block_data[thid_block] = d_degrees[thid]; // Assign value to block_data

//         __syncthreads();

//         int os = 1;

//         // Compute prefix sum
//         for (int d = blockDim.x >> 1; d > 0; d >>= 1) {
//             __syncthreads();
//             if (thid_block < d) {
//                 int ai = os * (2 * thid_block+1) - 1;
//                 int bi = os * (2 * thid_block+2) - 1;
//                 block_data[bi] += block_data[ai];
//             }

//             os *= 2;
//         }

//         if (thid_block == 0) { 
//             // result.block_sum = block_data[blockDim.x - 1];
//             block_data[blockDim.x - 1] = 0; // Clear the last element
//         }

//         for (int d = 1; d < blockDim.x; d *= 2) {

//             os /= 2;

//             __syncthreads();
//             if (thid_block < d) {
//                 int ai = os * (2 * thid_block+1) - 1;
//                 int bi = os * (2 * thid_block+2) - 1;
//                 int t = block_data[ai];

//                 block_data[ai] = block_data[bi];
//                 block_data[bi] += t;
//             }
//         }

//         __syncthreads();

//         d_degrees[thid] = block_data[thid_block];
//         // incrDegrees[thid] = block_data[blockDim.x - 1];

//         // printf("thid; scan; total %d %d %d\n", thid, d_degrees[thid], incrDegrees[thid]);

//     }
// }

// __global__ void prescan(long long int *g_odata, long long int *g_idata, int n) {
//     extern __shared__ float temp[]; // allocated on invocation
//     int thid = threadIdx.x;
//     int offset = 1;
    
//     // Load input into shared memory
//     temp[2 * thid] = g_idata[2 * thid];
//     temp[2 * thid + 1] = g_idata[2 * thid + 1];

//     // Build sum in place up the tree
//     for (int d = n >> 1; d > 0; d >>= 1) {
//         __syncthreads();
//         if (thid < d) {
//             int ai = offset * (2 * thid + 1) - 1;
//             int bi = offset * (2 * thid + 2) - 1;
//             temp[bi] += temp[ai];
//         }
//         offset *= 2;
//     }

//     // Clear the last element
//     if (thid == 0) {
//         temp[n - 1] = 0;
//     }

//     // Traverse down tree & build scan
//     for (int d = 1; d < n; d *= 2) {
//         offset >>= 1;
//         __syncthreads();
//         if (thid < d) {
//             int ai = offset * (2 * thid + 1) - 1;
//             int bi = offset * (2 * thid + 2) - 1;
//             int t = temp[ai];
//             temp[ai] = temp[bi];
//             temp[bi] += t;
//         }
//     }

//     __syncthreads();
//     // Write results to device memory
//     g_odata[2 * thid] = temp[2 * thid];
//     g_odata[2 * thid + 1] = temp[2 * thid + 1];
// }



__global__
void gather(int *d_adjacencyList, int *d_edgesOffset, int *d_parent, int queueSize,
            int *d_currentQueue, int *d_nextQueue, long long int *d_degrees, long long int *incrDegrees, int *d_distance, int iteration)
{
    int thid = blockIdx.x * blockDim.x + threadIdx.x;

    if (thid < queueSize) {
        // __shared__ int sharedIncrement;
        // if (!threadIdx.x) {
        //     sharedIncrement = incrDegrees[thid >> 10];
        // }
        // __syncthreads();

        // int sum = 0;
        // if (threadIdx.x) {
        //     sum = d_degrees[thid - 1];
        // }

        int nextQueuePlace = d_degrees[thid];

        int cur_node = d_currentQueue[thid];
        int row_offset_start = d_edgesOffset[cur_node];
        int row_offset_end = d_edgesOffset[cur_node+1];
        // int counter = 0;
        for (int i = row_offset_start; i < row_offset_end; i++)
        {
            int v = d_adjacencyList[i];
            if (d_distance[v] == -1) {
                // int nextQueuePlace = sharedIncrement + sum + counter;
                // printf("individiual %d %d\n", thid, nextQueuePlace);
                d_distance[v] = iteration + 1;
                d_nextQueue[nextQueuePlace] = v;
                nextQueuePlace++;
            }
        }
    }
}

__global__
void init_distance_kernel(int *device_distance, int *device_parent, int size, int source)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size)
    {
        if (i == source)
        {
            device_distance[i] = 0;
            device_parent[i] = 0;
        }
        else
        {
            device_distance[i] = -1;
            device_parent[i] = -1;
        }

    }
}

__global__ 
void intToLongLong(const int* src, int* dst, int num_elements) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < num_elements) {
        dst[idx] = static_cast<int>(src[idx]);
        // printf("thid and node %d %d\n", idx, src[idx]);
    }
}


__host__
gpuBFS::gpuBFS(csr &graph, int source)
{
    // std::cout << "hi1" << std::endl;

    hipMalloc(&d_col_idx, graph.num_edges * sizeof(int));
    hipMalloc(&d_row_offset, (graph.num_nodes+1) * sizeof(int));
    // hipMalloc(&d_edges_size, graph.num_nodes * sizeof(int));
    hipMalloc(&d_distance, graph.num_nodes * sizeof(int));
    hipMalloc(&d_parent, graph.num_nodes * sizeof(int));
    hipMalloc(&d_in_q, graph.num_nodes * sizeof(int));
    hipMalloc(&d_out_q, graph.num_nodes * sizeof(int));
    hipHostMalloc(&d_degrees, graph.num_nodes * sizeof(long long int));

    hipHostMalloc((void **) &d_degrees_total, graph.num_nodes * sizeof(long long int));

    // std::cout << "hi2" << std::endl;

    init_distance(graph, source);

    // std::cout << "hi3" << std::endl;

    int firstElementQueue = source;
    hipMemcpy(d_in_q, &firstElementQueue, sizeof(int), hipMemcpyHostToDevice);

    // std::cout << "hi4" << std::endl;
    
    // d_degrees_total = (int *)malloc(graph.num_nodes * sizeof(int));

    // intToLongLong<<<1024, graph.num_edges>>>(graph.col_idx, d_col_idx, graph.num_edges);
    // intToLongLong<<<1024, (graph.num_nodes+1)>>>(graph.row_offset, d_row_offset, (graph.num_nodes+1));
    hipMemcpy(d_col_idx, graph.col_idx, graph.num_edges * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_row_offset, graph.row_offset, (graph.num_nodes+1) * sizeof(int), hipMemcpyHostToDevice);

    // std::cout << "hi5" << std::endl;

    hipEvent_t gpu_start, gpu_end;
    hipEventCreate(&gpu_start);
    hipEventCreate(&gpu_end);
    hipEventRecord(gpu_start);

    int *queueSize;
    queueSize = (int *)malloc(sizeof(int));
    *queueSize = 1;
    int *nextQueueSize;
    nextQueueSize = (int *)malloc(sizeof(int));
    *nextQueueSize = 0;
    iteration = 0;
    while (*queueSize)
    {

        std::cout << "iter and size: " << iteration << " " << *queueSize << std::endl;
        
        // next layer phase
        int block_size = 1024;
        int num_blocks = *queueSize / block_size + 1;

        countDegrees<<<num_blocks, block_size>>>(d_col_idx, d_row_offset, d_parent, *queueSize, d_in_q, d_degrees, d_distance);
        hipDeviceSynchronize();
        
        prescan_large_unoptimized<<<num_blocks, block_size>>>(d_degrees_total, d_degrees, 1024, d_sums);
        // block_prefix_sum<<<num_blocks, block_size>>>(*queueSize, d_degrees) ;
        // prescan<<<num_blocks, block_size>>>(d_degrees_total, d_degrees, *queueSize) ;
        hipDeviceSynchronize();

        // *nextQueueSize = d_degrees_total[(*queueSize - 1) / 1024 + 1];
        // std::cout << *queueSize << std::endl;
        // *nextQueueSize = d_degrees_total[*queueSize-1];
        // std::cout << d_degrees[*queueSize-1] << std::endl;
        *nextQueueSize = d_degrees[*queueSize];
        std::cout << *nextQueueSize << std::endl;

        gather<<<num_blocks, block_size>>>(d_col_idx, d_row_offset, d_parent, *queueSize, d_in_q, d_out_q, d_degrees_total, d_degrees_total, d_distance, iteration);
        hipDeviceSynchronize();

        iteration++;
        *queueSize = *nextQueueSize;
        std::swap(d_in_q, d_out_q);

        if (iteration == 4)
        {
            break;
        }

        // break;
    }

    hipMemcpy(host_distance, d_distance, graph.num_nodes * sizeof(int), hipMemcpyDeviceToHost);
    // std::cout << host_distance[1] << " " << host_distance[2] << std::endl;
    // host_distance[source] = 0;

    hipEventRecord(gpu_end);
    hipEventSynchronize(gpu_end);
    hipEventElapsedTime(&exec_time, gpu_start, gpu_end);

    free(queueSize);
    free(nextQueueSize);

}

__host__
void gpuBFS::init_distance(csr &graph, int source)
{
    // allocate host memory
    host_distance = (int *)malloc(graph.num_nodes * sizeof(int));

    // allocate device memory
    hipMalloc(&d_distance, graph.num_nodes * sizeof(int));

    // copy memory from host to device
    hipMemcpy(d_distance, host_distance, graph.num_nodes * sizeof(int), hipMemcpyHostToDevice);

    // run kernel to inialize distance
    dim3 block(1024, 1);
    dim3 grid((graph.num_nodes+block.x-1)/block.x, 1);
    init_distance_kernel<<<grid, block>>>(d_distance, d_parent, graph.num_nodes, source);

    // copy back
    hipMemcpy(host_distance, d_distance, graph.num_nodes * sizeof(int), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
}


__host__
gpuBFS::~gpuBFS()
{
    free(host_distance);
    // free(host_queue);
    

    hipFree(d_distance);
    hipFree(d_in_q);
    hipFree(d_out_q);
    hipFree(d_parent);
    hipFree(d_degrees);
    hipFree(d_col_idx);
    hipFree(d_row_offset);
    hipFree(d_degrees_total);
}
